#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common/include/dumping_functions.h"
#include "common/include/forward_functions.h"
#include "hashtable/simple_hashtable.h"
#include "operation/operation_interface.h"

namespace SparseOperationKit {

template <typename EmbeddingType>
__global__ static void gatherKernel(const size_t EmbeddingDimension, EmbeddingType *inputs,
                                    size_t *indices, size_t num_indices, EmbeddingType *outputs) {
  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < num_indices * EmbeddingDimension;
       id += blockDim.x * gridDim.x) {
    size_t item_id = id / EmbeddingDimension;
    size_t embedding_id = id - item_id * EmbeddingDimension;

    size_t index = static_cast<size_t>(indices[item_id]);
    outputs[id] = inputs[index * EmbeddingDimension + embedding_id];
  }
}

class DenseGather : public EmbeddingLookuper {
 public:
  DenseGather(ConstructionContext_t context, std::shared_ptr<ParamInterface> param)
      : EmbeddingLookuper(context, param),
        resource_mgr_(base_context()->get_resource_mgr()),
        num_keys_per_rank_(base_context()->get_replica_batch_size() *
                           base_context()->get_slot_num() * base_context()->get_nnz_per_slot()) {
    const size_t local_gpu_count = resource_mgr_->get_local_gpu_count();
    mapped_indices_buf_.reserve(local_gpu_count);
    host_nnz_.reserve(local_gpu_count);
    gathered_embeddings_buf_.reserve(local_gpu_count);

    if (sizeof(size_t) != sizeof(int64_t))
      throw std::runtime_error(
          "In this platform, sizeof(size_t) != sizeof(int64_t). "
          "It will cause unexpected behavoir when copy datas from "
          "size_t pointer to int64_t pointer.");

    if (param->get_hashtable(0)->identical_mapping()) {
      // identical_mapping waste memory spaces, so that lookuper
      // will set its wanted hashtable for param
      const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
      auto stream = resource_mgr_->get_local_gpu(0)->get_stream();
      const size_t capacity = param->get_hashtable(0)->get_capacity(stream);
      HashFunctor_t hash_func = HashFunctors::Divisive<int64_t, size_t>::create(
          /*interval=*/global_gpu_count, /*capacity=*/capacity,
          /*global_replica_id=*/resource_mgr_->cal_global_id_from_local_id(0));
      auto hashtable = SimpleHashtable<int64_t, size_t>::create(capacity, hash_func);
      param->set_hashtable(hashtable);
    }  // if identical_mapping
  }

  void allocate_forward_spaces() override {
    const size_t local_gpu_count = resource_mgr_->get_local_gpu_count();
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t embedding_vec_size = base_context()->get_param()->get_embedding_vec_size();
    for (size_t dev_id = 0; dev_id < local_gpu_count; dev_id++) {
      auto &buffer = base_context()->get_buffer(dev_id);
      auto &host_buffer = base_context()->get_host_buffer(dev_id);
      {
        Tensor2<size_t> tensor;
        buffer->reserve({global_gpu_count, num_keys_per_rank_}, &tensor);
        mapped_indices_buf_.push_back(tensor);
      }
      {
        Tensor2<float> tensor;
        buffer->reserve({global_gpu_count, embedding_vec_size * num_keys_per_rank_}, &tensor);
        gathered_embeddings_buf_.push_back(tensor);
      }
      {
        Tensor2<size_t> tensor;
        host_buffer->reserve({1}, &tensor);
        host_nnz_.push_back(tensor);
      }
    }  // for dev_id in local_gpu_count
  }
  void allocate_backward_spaces() override {}
  void forward(const Context_t &replica_context, const bool training) override {
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t global_replica_id = replica_context->get_global_replica_id();
    const size_t local_replica_id = resource_mgr_->cal_local_id_from_global_id(global_replica_id);
    const auto &local_gpu = resource_mgr_->get_local_gpu(local_replica_id);

    auto &hashtable = param_->get_hashtable(local_replica_id);

    const auto &replica_exchanged_keys = replica_context->input("replica_exchanged_keys");
    const auto &replica_h_recv_chunk_offsets =
        replica_context->input("replica_h_recv_chunk_offsets");
    const uint32_t h_local_nnz =
        replica_h_recv_chunk_offsets->GetPtrWithType<uint32_t>()[global_gpu_count];
    // step 1: get index using keys
    if (training) {
      hashtable->get_insert(replica_exchanged_keys->GetPtrWithType<int64_t>(),
                            mapped_indices_buf_[local_replica_id].get_ptr(),
                            /*nnz=*/h_local_nnz, local_gpu->get_stream());
    } else {
      hashtable->get(replica_exchanged_keys->GetPtrWithType<int64_t>(),
                     mapped_indices_buf_[local_replica_id].get_ptr(),
                     /*nnz=*/h_local_nnz, local_gpu->get_stream());
    }

    // step 2: gather embedding vectors from embedding table
    const auto &embedding_table = param_->get_embedding_table_tensor(local_replica_id);
    gatherKernel<float><<<local_gpu->get_sm_count() * 2, 1024ul, 0, local_gpu->get_stream()>>>(
        /*EmbeddingDimension=*/param_->get_embedding_vec_size(),
        /*inputs=*/embedding_table->GetPtrWithType<float>(),
        /*indices=*/mapped_indices_buf_[local_replica_id].get_ptr(),
        /*num_indices=*/h_local_nnz,
        /*outputs=*/gathered_embeddings_buf_[local_replica_id].get_ptr());
    CK_CUDA(hipGetLastError());

    // step 3: set the output of embedding lookuper
    replica_context->set_output("replica_gathered_embeddings",
                                gathered_embeddings_buf_[local_replica_id]);
    // write host_nnz in current iteration
    host_nnz_[local_replica_id].get_ptr()[0] = static_cast<size_t>(h_local_nnz);
    replica_context->set_output("replica_host_nnz", host_nnz_[local_replica_id]);
  }

  void backward(const Context_t &replica_context) override {
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t global_replica_id = replica_context->get_global_replica_id();
    const size_t local_replica_id = resource_mgr_->cal_local_id_from_global_id(global_replica_id);
    const auto &local_gpu = resource_mgr_->get_local_gpu(local_replica_id);

    const auto &replica_h_recv_chunk_offsets =
        replica_context->input("replica_h_recv_chunk_offsets");
    const uint32_t h_local_nnz =
        replica_h_recv_chunk_offsets->GetPtrWithType<uint32_t>()[global_gpu_count];
    auto &replica_value_index_tensor = replica_context->output("value_index_tensor");

    // FIXME: what if sizeof(size_t) != sizeof(int64_t)
    CK_CUDA(hipMemcpyAsync(replica_value_index_tensor->GetPtrWithType<int64_t>(),
                            mapped_indices_buf_[local_replica_id].get_ptr(),
                            sizeof(size_t) * h_local_nnz, hipMemcpyDeviceToDevice,
                            local_gpu->get_stream()));
  }

  void save_params(std::shared_ptr<Tensor> &keys, std::shared_ptr<Tensor> &embedding_values,
                   size_t &num_total_keys) const override {
    // this lookuper distribute keys to each GPU based on key % GPU_NUM
    save_params_helper(param_, resource_mgr_, keys, embedding_values, num_total_keys);
  }

  void restore_params(const std::shared_ptr<Tensor> &keys,
                      const std::shared_ptr<Tensor> &embedding_values,
                      const size_t num_total_keys) override {
    // this lookuper distribute keys to each GPU based on key % GPU_NUM
    restore_params_helper(param_, resource_mgr_, keys, embedding_values, num_total_keys);
  }

 private:
  std::shared_ptr<ResourcesManager> resource_mgr_;
  const size_t num_keys_per_rank_;

  // forward spaces
  Tensors2<size_t> mapped_indices_buf_;
  Tensors2<size_t> host_nnz_;
  Tensors2<float> gathered_embeddings_buf_;
};

REGISTER_EMB_LOOKUPER_BUILDER("dense_gather", DenseGather);

}  // namespace SparseOperationKit