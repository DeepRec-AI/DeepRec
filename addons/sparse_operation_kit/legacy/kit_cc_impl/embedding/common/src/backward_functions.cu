#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cmath>

#include "common/include/backward_functions.cuh"
#include "common/include/backward_functions.h"

namespace SparseOperationKit {

template <typename TypeKey>
__global__ void expand_input_grad_kernel(const size_t global_batch_size, const size_t slot_num,
                                         const size_t embedding_vec_size,
                                         const TypeKey *replica_row_offset, const float *wgrad,
                                         float *replica_input_grad) {
  size_t bid = blockIdx.x;   // each block corresponding to one sample
  size_t tid = threadIdx.x;  // each thread corresponding to one element in the wgrad

  if (bid < global_batch_size && tid < embedding_vec_size) {
    for (size_t i = 0; i < slot_num; i++) {
      size_t row_index = bid * slot_num + i;  // row-index in wgrad
      TypeKey value_offset = replica_row_offset[row_index];
      TypeKey feature_num = replica_row_offset[row_index + 1] - value_offset;

      // in a slot
      for (size_t j = 0; j < feature_num; j++) {
        size_t target_index = value_offset + j;
        replica_input_grad[target_index * embedding_vec_size + tid] =
            wgrad[row_index * embedding_vec_size + tid];
      }
    }  // for i in slot_num
  }    // if bid < global_batch_size && tid < embedding_vec_size
}

template <typename TypeKey>
void expand_input_grad(const size_t global_batch_size, const size_t slot_num,
                       const size_t embedding_vec_size, const TypeKey *replica_row_offset,
                       const float *wgrad, float *replica_input_grad, hipStream_t stream) {
  const size_t grid_size = global_batch_size;
  const size_t block_size = embedding_vec_size;
  expand_input_grad_kernel<<<grid_size, block_size, 0, stream>>>(
      global_batch_size, slot_num, embedding_vec_size, replica_row_offset, wgrad,
      replica_input_grad);
}

template void expand_input_grad(const size_t global_batch_size, const size_t slot_num,
                                const size_t embedding_vec_size, const int64_t *replica_row_offset,
                                const float *wgrad, float *replica_input_grad, hipStream_t stream);

template <typename TypeEmbeddingComp>
void backward_sum(size_t batch_size, size_t slot_num, size_t embedding_vec_size,
                  const TypeEmbeddingComp *top_grad, TypeEmbeddingComp *wgrad,
                  hipStream_t stream) {
  HugeCTR::backward_sum(batch_size, slot_num, embedding_vec_size, top_grad, wgrad, stream);
}

template void backward_sum(size_t batch_size, size_t slot_num, size_t embedding_vec_size,
                           const float *top_grad, float *wgrad, hipStream_t stream);

template <typename TypeKey, typename TypeEmbeddingComp>
void backward_mean(size_t batch_size, size_t slot_size, size_t embedding_vec_size,
                   const TypeKey *row_offset, const TypeEmbeddingComp *top_grad,
                   TypeEmbeddingComp *wgrad, hipStream_t stream) {
  HugeCTR::backward_mean(batch_size, slot_size, embedding_vec_size, row_offset, top_grad, wgrad,
                         stream);
}

template void backward_mean(size_t batch_size, size_t slot_size, size_t embedding_vec_size,
                            const int64_t *row_offset, const float *top_grad, float *wgrad,
                            hipStream_t stream);

}  // namespace SparseOperationKit
